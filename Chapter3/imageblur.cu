#include <iostream>
#include <hip/hip_runtime.h>
#include <fstream>

const int BLUR_SIZE = 2;

__global__ void blurKernel(unsigned char *in, unsigned char *out, int w, int h) {
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;
    
    if(col < w && row < h) {
        int pixVal = 0;
        int pixels = 0;
        
        for(int blurRow=-BLUR_SIZE; blurRow<BLUR_SIZE+1; ++blurRow) {
            for(int blurCol=-BLUR_SIZE; blurCol<BLUR_SIZE+1; ++blurCol) {
                int curRow = row + blurRow;
                int curCol = col + blurCol;
                
                if(curRow>=0 && curRow<h && curCol>=0 && curCol<w) {
                    pixVal += in[curRow*w + curCol];
                    ++pixels;
                }
            }
        }
        out[row*w + col] = (unsigned char)(pixVal/pixels);
    }
}

int main() {
    // Create test image (grayscale gradient)
    const int width = 256;
    const int height = 256;
    unsigned char* h_in = new unsigned char[width * height];
    unsigned char* h_out = new unsigned char[width * height];
    
    for(int i = 0; i < height; i++) {
        for(int j = 0; j < width; j++) {
            h_in[i*width + j] = (i + j) % 256;  // Create gradient pattern
        }
    }

    // Allocate device memory
    unsigned char *d_in, *d_out;
    hipMalloc(&d_in, width * height * sizeof(unsigned char));
    hipMalloc(&d_out, width * height * sizeof(unsigned char));
    
    // Copy input to device
    hipMemcpy(d_in, h_in, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);
    
    // Launch kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, 
                 (height + blockDim.y - 1) / blockDim.y);
    blurKernel<<<gridDim, blockDim>>>(d_in, d_out, width, height);
    
    // Copy result back
    hipMemcpy(h_out, d_out, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // Save input and output as raw files for comparison
    std::ofstream fout_in("input.raw", std::ios::binary);
    fout_in.write((char*)h_in, width * height);
    fout_in.close();

    std::ofstream fout_out("output.raw", std::ios::binary);
    fout_out.write((char*)h_out, width * height);
    fout_out.close();

    // Cleanup
    delete[] h_in;
    delete[] h_out;
    hipFree(d_in);
    hipFree(d_out);
    
    std::cout << "Files saved: input.raw and output.raw\n";
    return 0;
}
